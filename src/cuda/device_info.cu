#include "binstatcuda/device_info.cuh"

#include <hip/hip_runtime.h>

namespace binstatcuda {

int cuda_device_count() noexcept {
    int count = 0;
    const hipError_t status = hipGetDeviceCount(&count);
    if (status != hipSuccess) {
        return -1;
    }
    return count;
}

}  // namespace binstatcuda
