#include "hip/hip_runtime.h"
#include "binstatcuda/histogram.cuh"

#include <algorithm>
#include <cstddef>
#include <cstdint>

#include <hip/hip_runtime.h>

namespace binstatcuda {
namespace {

constexpr int THREADS_PER_BLOCK = 256;
constexpr std::size_t MAX_SHARED_BINS = 4096;

template <typename T>
class DeviceBuffer {
public:
    DeviceBuffer() = default;
    DeviceBuffer(const DeviceBuffer&) = delete;
    DeviceBuffer& operator=(const DeviceBuffer&) = delete;

    ~DeviceBuffer() noexcept {
        reset();
    }

    hipError_t allocate(std::size_t count) noexcept {
        reset();
        if (count == 0) {
            return hipSuccess;
        }
        T* ptr = nullptr;
        const hipError_t status = hipMalloc(
            reinterpret_cast<void**>(&ptr), count * sizeof(T)
        );
        if (status == hipSuccess) {
            ptr_ = ptr;
            size_ = count;
        }
        return status;
    }

    void reset() noexcept {
        if (ptr_ != nullptr) {
            hipFree(ptr_);
            ptr_ = nullptr;
            size_ = 0;
        }
    }

    [[nodiscard]] T* get() noexcept {
        return ptr_;
    }

    [[nodiscard]] const T* get() const noexcept {
        return ptr_;
    }

private:
    T* ptr_ = nullptr;
    std::size_t size_ = 0;
};

__device__ __forceinline__ int find_bin(
    float value,
    const float* edges,
    int edge_count
) noexcept {
    if (value < edges[0] || value > edges[edge_count - 1]) {
        return -1;
    }

    int left = 0;
    int right = edge_count - 1;

    while (left < right) {
        const int mid = (left + right) / 2;
        if (value >= edges[mid]) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }

    int bin = left - 1;
    if (bin < 0) {
        return -1;
    }

    if (value == edges[edge_count - 1]) {
        bin = edge_count - 2;
    }

    return bin;
}

__global__ void histogram_1d_global_kernel(
    const float* samples,
    std::size_t sample_count,
    const float* edges,
    int edge_count,
    unsigned long long* counts
) {
    const std::size_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
    const std::size_t stride = blockDim.x * gridDim.x;

    for (std::size_t idx = global_id; idx < sample_count; idx += stride) {
        const float value = samples[idx];
        const int bin = find_bin(value, edges, edge_count);
        if (bin >= 0) {
            atomicAdd(&counts[bin], 1ULL);
        }
    }
}

__global__ void histogram_1d_shared_kernel(
    const float* samples,
    std::size_t sample_count,
    const float* edges,
    int edge_count,
    unsigned long long* counts
) {
    extern __shared__ unsigned long long shared_counts[];

    const int bin_count = edge_count - 1;

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        shared_counts[idx] = 0ULL;
    }
    __syncthreads();

    const std::size_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
    const std::size_t stride = blockDim.x * gridDim.x;

    for (std::size_t sample_idx = global_id;
         sample_idx < sample_count;
         sample_idx += stride) {
        const float value = samples[sample_idx];
        const int bin = find_bin(value, edges, edge_count);
        if (bin >= 0) {
            atomicAdd(&shared_counts[bin], 1ULL);
        }
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        const unsigned long long value = shared_counts[idx];
        if (value != 0ULL) {
            atomicAdd(&counts[idx], value);
        }
    }
}

__global__ void histogram_2d_global_kernel(
    const float* xs,
    const float* ys,
    std::size_t sample_count,
    const float* x_edges,
    int x_edge_count,
    const float* y_edges,
    int y_edge_count,
    unsigned long long* counts
) {
    const std::size_t global_id =
        static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    const std::size_t stride =
        static_cast<std::size_t>(blockDim.x) * gridDim.x;

    const int y_bins = y_edge_count - 1;

    for (std::size_t idx = global_id; idx < sample_count; idx += stride) {
        const int bin_x = find_bin(xs[idx], x_edges, x_edge_count);
        if (bin_x < 0) {
            continue;
        }
        const int bin_y = find_bin(ys[idx], y_edges, y_edge_count);
        if (bin_y < 0) {
            continue;
        }
        const std::size_t offset =
            static_cast<std::size_t>(bin_x)
            * static_cast<std::size_t>(y_bins)
            + static_cast<std::size_t>(bin_y);
        atomicAdd(&counts[offset], 1ULL);
    }
}

__global__ void histogram_2d_shared_kernel(
    const float* xs,
    const float* ys,
    std::size_t sample_count,
    const float* x_edges,
    int x_edge_count,
    const float* y_edges,
    int y_edge_count,
    unsigned long long* counts
) {
    const int x_bins = x_edge_count - 1;
    const int y_bins = y_edge_count - 1;
    const int bin_count = x_bins * y_bins;

    extern __shared__ unsigned long long shared_counts[];

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        shared_counts[idx] = 0ULL;
    }
    __syncthreads();

    const std::size_t global_id =
        static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    const std::size_t stride =
        static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (std::size_t sample_idx = global_id;
         sample_idx < sample_count;
         sample_idx += stride) {
        const int bin_x = find_bin(xs[sample_idx], x_edges, x_edge_count);
        if (bin_x < 0) {
            continue;
        }
        const int bin_y = find_bin(ys[sample_idx], y_edges, y_edge_count);
        if (bin_y < 0) {
            continue;
        }
        const std::size_t offset =
            static_cast<std::size_t>(bin_x)
            * static_cast<std::size_t>(y_bins)
            + static_cast<std::size_t>(bin_y);
        atomicAdd(&shared_counts[offset], 1ULL);
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        const unsigned long long value = shared_counts[idx];
        if (value != 0ULL) {
            atomicAdd(&counts[idx], value);
        }
    }
}

__global__ void binned_statistic_1d_global_kernel(
    const float* samples,
    const float* values,
    std::size_t sample_count,
    const float* edges,
    int edge_count,
    unsigned long long* counts,
    float* sums,
    unsigned int* bin_numbers
) {
    const std::size_t global_id =
        static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    const std::size_t stride =
        static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (std::size_t idx = global_id; idx < sample_count; idx += stride) {
        const int bin = find_bin(samples[idx], edges, edge_count);
        const unsigned int bin_number =
            bin >= 0 ? static_cast<unsigned int>(bin + 1) : 0U;
        bin_numbers[idx] = bin_number;
        if (bin < 0) {
            continue;
        }

        atomicAdd(&counts[bin], 1ULL);
        atomicAdd(&sums[bin], values[idx]);
    }
}

__global__ void binned_statistic_1d_shared_kernel(
    const float* samples,
    const float* values,
    std::size_t sample_count,
    const float* edges,
    int edge_count,
    unsigned long long* counts,
    float* sums,
    unsigned int* bin_numbers
) {
    const int bin_count = edge_count - 1;
    extern __shared__ unsigned char shared_storage[];
    auto* shared_counts =
        reinterpret_cast<unsigned long long*>(shared_storage);
    float* shared_sums =
        reinterpret_cast<float*>(shared_counts + bin_count);

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        shared_counts[idx] = 0ULL;
        shared_sums[idx] = 0.0F;
    }
    __syncthreads();

    const std::size_t global_id =
        static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    const std::size_t stride =
        static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (std::size_t sample_idx = global_id;
         sample_idx < sample_count;
         sample_idx += stride) {
        const int bin = find_bin(samples[sample_idx], edges, edge_count);
        const unsigned int bin_number =
            bin >= 0 ? static_cast<unsigned int>(bin + 1) : 0U;
        bin_numbers[sample_idx] = bin_number;
        if (bin < 0) {
            continue;
        }

        atomicAdd(&shared_counts[bin], 1ULL);
        atomicAdd(&shared_sums[bin], values[sample_idx]);
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        atomicAdd(&counts[idx], shared_counts[idx]);
        atomicAdd(&sums[idx], shared_sums[idx]);
    }
}

__global__ void binned_statistic_2d_global_kernel(
    const float* xs,
    const float* ys,
    const float* values,
    std::size_t sample_count,
    const float* x_edges,
    int x_edge_count,
    const float* y_edges,
    int y_edge_count,
    unsigned long long* counts,
    float* sums,
    unsigned int* bin_numbers_x,
    unsigned int* bin_numbers_y
) {
    const std::size_t global_id =
        static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    const std::size_t stride =
        static_cast<std::size_t>(blockDim.x) * gridDim.x;

    const int y_bins = y_edge_count - 1;

    for (std::size_t idx = global_id; idx < sample_count; idx += stride) {
        const int bin_x = find_bin(xs[idx], x_edges, x_edge_count);
        const int bin_y = find_bin(ys[idx], y_edges, y_edge_count);

        const unsigned int bin_number_x =
            bin_x >= 0 ? static_cast<unsigned int>(bin_x + 1) : 0U;
        const unsigned int bin_number_y =
            bin_y >= 0 ? static_cast<unsigned int>(bin_y + 1) : 0U;
        bin_numbers_x[idx] = bin_number_x;
        bin_numbers_y[idx] = bin_number_y;

        if (bin_x < 0 || bin_y < 0) {
            continue;
        }

        const std::size_t offset =
            static_cast<std::size_t>(bin_x)
            * static_cast<std::size_t>(y_bins)
            + static_cast<std::size_t>(bin_y);

        atomicAdd(&counts[offset], 1ULL);
        atomicAdd(&sums[offset], values[idx]);
    }
}

__global__ void binned_statistic_2d_shared_kernel(
    const float* xs,
    const float* ys,
    const float* values,
    std::size_t sample_count,
    const float* x_edges,
    int x_edge_count,
    const float* y_edges,
    int y_edge_count,
    unsigned long long* counts,
    float* sums,
    unsigned int* bin_numbers_x,
    unsigned int* bin_numbers_y
) {
    const int x_bins = x_edge_count - 1;
    const int y_bins = y_edge_count - 1;
    const int bin_count = x_bins * y_bins;

    extern __shared__ unsigned char shared_storage[];
    auto* shared_counts =
        reinterpret_cast<unsigned long long*>(shared_storage);
    float* shared_sums =
        reinterpret_cast<float*>(shared_counts + bin_count);

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        shared_counts[idx] = 0ULL;
        shared_sums[idx] = 0.0F;
    }
    __syncthreads();

    const std::size_t global_id =
        static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    const std::size_t stride =
        static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (std::size_t sample_idx = global_id;
         sample_idx < sample_count;
         sample_idx += stride) {
        const int bin_x = find_bin(xs[sample_idx], x_edges, x_edge_count);
        const int bin_y = find_bin(ys[sample_idx], y_edges, y_edge_count);

        const unsigned int bin_number_x =
            bin_x >= 0 ? static_cast<unsigned int>(bin_x + 1) : 0U;
        const unsigned int bin_number_y =
            bin_y >= 0 ? static_cast<unsigned int>(bin_y + 1) : 0U;
        bin_numbers_x[sample_idx] = bin_number_x;
        bin_numbers_y[sample_idx] = bin_number_y;

        if (bin_x < 0 || bin_y < 0) {
            continue;
        }

        const std::size_t offset =
            static_cast<std::size_t>(bin_x)
            * static_cast<std::size_t>(y_bins)
            + static_cast<std::size_t>(bin_y);

        atomicAdd(&shared_counts[offset], 1ULL);
        atomicAdd(&shared_sums[offset], values[sample_idx]);
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < bin_count; idx += blockDim.x) {
        atomicAdd(&counts[idx], shared_counts[idx]);
        atomicAdd(&sums[idx], shared_sums[idx]);
    }
}

[[nodiscard]] int compute_grid_size(std::size_t sample_count) noexcept {
    if (sample_count == 0) {
        return 1;
    }
    const std::size_t blocks =
        (sample_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    return static_cast<int>(std::min<std::size_t>(blocks, 65535));
}

[[nodiscard]] hipError_t launch_histogram_1d(
    const float* d_samples,
    std::size_t sample_count,
    const float* d_edges,
    int edge_count,
    unsigned long long* d_counts
) noexcept {
    const int grid_size = compute_grid_size(sample_count);

    const int bin_count = edge_count - 1;
    const bool use_shared =
        static_cast<std::size_t>(bin_count) <= MAX_SHARED_BINS;

    if (use_shared) {
        const std::size_t shared_bytes =
            static_cast<std::size_t>(bin_count)
            * sizeof(unsigned long long);
        histogram_1d_shared_kernel<<<grid_size, THREADS_PER_BLOCK, shared_bytes>>>(
            d_samples,
            sample_count,
            d_edges,
            edge_count,
            d_counts
        );
    } else {
        histogram_1d_global_kernel<<<grid_size, THREADS_PER_BLOCK>>>(
            d_samples,
            sample_count,
            d_edges,
            edge_count,
            d_counts
        );
    }

    hipError_t status = hipGetLastError();
    if (status != hipSuccess) {
        return status;
    }

    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

[[nodiscard]] hipError_t launch_histogram_2d(
    const float* d_xs,
    const float* d_ys,
    std::size_t sample_count,
    const float* d_x_edges,
    int x_edge_count,
    const float* d_y_edges,
    int y_edge_count,
    unsigned long long* d_counts
) noexcept {
    const int grid_size = compute_grid_size(sample_count);

    const int x_bins = x_edge_count - 1;
    const int y_bins = y_edge_count - 1;
    const std::size_t bin_count =
        static_cast<std::size_t>(x_bins) * static_cast<std::size_t>(y_bins);
    const bool use_shared = bin_count <= MAX_SHARED_BINS;

    if (use_shared) {
        const std::size_t shared_bytes =
            bin_count * sizeof(unsigned long long);
        histogram_2d_shared_kernel<<<grid_size, THREADS_PER_BLOCK, shared_bytes>>>(
            d_xs,
            d_ys,
            sample_count,
            d_x_edges,
            x_edge_count,
            d_y_edges,
            y_edge_count,
            d_counts
        );
    } else {
        histogram_2d_global_kernel<<<grid_size, THREADS_PER_BLOCK>>>(
            d_xs,
            d_ys,
            sample_count,
            d_x_edges,
            x_edge_count,
            d_y_edges,
            y_edge_count,
            d_counts
        );
    }

    hipError_t status = hipGetLastError();
    if (status != hipSuccess) {
        return status;
    }

    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

[[nodiscard]] hipError_t launch_binned_statistic_1d(
    const float* d_samples,
    const float* d_values,
    std::size_t sample_count,
    const float* d_edges,
    int edge_count,
    unsigned long long* d_counts,
    float* d_sums,
    unsigned int* d_bin_numbers
) noexcept {
    const int grid_size = compute_grid_size(sample_count);
    const int bin_count = edge_count - 1;
    const bool use_shared =
        static_cast<std::size_t>(bin_count) <= MAX_SHARED_BINS;

    if (use_shared) {
        const std::size_t shared_bytes =
            static_cast<std::size_t>(bin_count)
            * (sizeof(unsigned long long) + sizeof(float));
        binned_statistic_1d_shared_kernel<<<grid_size, THREADS_PER_BLOCK, shared_bytes>>>(
            d_samples,
            d_values,
            sample_count,
            d_edges,
            edge_count,
            d_counts,
            d_sums,
            d_bin_numbers
        );
    } else {
        binned_statistic_1d_global_kernel<<<grid_size, THREADS_PER_BLOCK>>>(
            d_samples,
            d_values,
            sample_count,
            d_edges,
            edge_count,
            d_counts,
            d_sums,
            d_bin_numbers
        );
    }

    hipError_t status = hipGetLastError();
    if (status != hipSuccess) {
        return status;
    }

    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

[[nodiscard]] hipError_t launch_binned_statistic_2d(
    const float* d_xs,
    const float* d_ys,
    const float* d_values,
    std::size_t sample_count,
    const float* d_x_edges,
    int x_edge_count,
    const float* d_y_edges,
    int y_edge_count,
    unsigned long long* d_counts,
    float* d_sums,
    unsigned int* d_bin_numbers_x,
    unsigned int* d_bin_numbers_y
) noexcept {
    const int grid_size = compute_grid_size(sample_count);

    const int x_bins = x_edge_count - 1;
    const int y_bins = y_edge_count - 1;
    const std::size_t bin_count =
        static_cast<std::size_t>(x_bins) * static_cast<std::size_t>(y_bins);
    const bool use_shared = bin_count <= MAX_SHARED_BINS;

    if (use_shared) {
        const std::size_t shared_bytes =
            bin_count * (sizeof(unsigned long long) + sizeof(float));
        binned_statistic_2d_shared_kernel<<<grid_size, THREADS_PER_BLOCK, shared_bytes>>>(
            d_xs,
            d_ys,
            d_values,
            sample_count,
            d_x_edges,
            x_edge_count,
            d_y_edges,
            y_edge_count,
            d_counts,
            d_sums,
            d_bin_numbers_x,
            d_bin_numbers_y
        );
    } else {
        binned_statistic_2d_global_kernel<<<grid_size, THREADS_PER_BLOCK>>>(
            d_xs,
            d_ys,
            d_values,
            sample_count,
            d_x_edges,
            x_edge_count,
            d_y_edges,
            y_edge_count,
            d_counts,
            d_sums,
            d_bin_numbers_x,
            d_bin_numbers_y
        );
    }

    hipError_t status = hipGetLastError();
    if (status != hipSuccess) {
        return status;
    }

    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

}  // namespace

hipError_t histogram_1d(
    const float* host_samples,
    std::size_t sample_count,
    const float* host_edges,
    int edge_count,
    unsigned long long* host_counts
) noexcept {
    if (edge_count < 2) {
        return hipErrorInvalidValue;
    }

    const int bin_count = edge_count - 1;
    std::fill(host_counts, host_counts + bin_count, 0ULL);

    if (sample_count == 0) {
        return hipSuccess;
    }

    DeviceBuffer<float> d_samples;
    DeviceBuffer<float> d_edges;
    DeviceBuffer<unsigned long long> d_counts;

    const hipError_t alloc_samples = d_samples.allocate(sample_count);
    if (alloc_samples != hipSuccess) {
        return alloc_samples;
    }

    const hipError_t alloc_edges = d_edges.allocate(edge_count);
    if (alloc_edges != hipSuccess) {
        return alloc_edges;
    }

    const hipError_t alloc_counts = d_counts.allocate(bin_count);
    if (alloc_counts != hipSuccess) {
        return alloc_counts;
    }

    hipError_t status = hipMemcpy(
        d_samples.get(),
        host_samples,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_edges.get(),
        host_edges,
        edge_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemset(
        d_counts.get(),
        0,
        static_cast<std::size_t>(bin_count) * sizeof(unsigned long long)
    );
    if (status != hipSuccess) {
        return status;
    }

    status = launch_histogram_1d(
        d_samples.get(),
        sample_count,
        d_edges.get(),
        edge_count,
        d_counts.get()
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_counts,
        d_counts.get(),
        static_cast<std::size_t>(bin_count) * sizeof(unsigned long long),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

hipError_t histogram_2d(
    const float* host_x,
    const float* host_y,
    std::size_t sample_count,
    const float* host_x_edges,
    int x_edge_count,
    const float* host_y_edges,
    int y_edge_count,
    unsigned long long* host_counts
) noexcept {
    if (x_edge_count < 2 || y_edge_count < 2) {
        return hipErrorInvalidValue;
    }

    const int x_bin_count = x_edge_count - 1;
    const int y_bin_count = y_edge_count - 1;
    const std::size_t total_bins =
        static_cast<std::size_t>(x_bin_count)
        * static_cast<std::size_t>(y_bin_count);
    std::fill(host_counts, host_counts + total_bins, 0ULL);

    if (sample_count == 0) {
        return hipSuccess;
    }

    DeviceBuffer<float> d_x;
    DeviceBuffer<float> d_y;
    DeviceBuffer<float> d_x_edges;
    DeviceBuffer<float> d_y_edges;
    DeviceBuffer<unsigned long long> d_counts;

    hipError_t status = d_x.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_y.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_x_edges.allocate(x_edge_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_y_edges.allocate(y_edge_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_counts.allocate(total_bins);
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_x.get(),
        host_x,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_y.get(),
        host_y,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_x_edges.get(),
        host_x_edges,
        x_edge_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_y_edges.get(),
        host_y_edges,
        y_edge_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemset(
        d_counts.get(),
        0,
        total_bins * sizeof(unsigned long long)
    );
    if (status != hipSuccess) {
        return status;
    }

    status = launch_histogram_2d(
        d_x.get(),
        d_y.get(),
        sample_count,
        d_x_edges.get(),
        x_edge_count,
        d_y_edges.get(),
        y_edge_count,
        d_counts.get()
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_counts,
        d_counts.get(),
        total_bins * sizeof(unsigned long long),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

hipError_t binned_statistic_1d(
    const float* host_samples,
    const float* host_values,
    std::size_t sample_count,
    const float* host_edges,
    int edge_count,
    unsigned long long* host_counts,
    float* host_sums,
    unsigned int* host_bin_numbers
) noexcept {
    if (edge_count < 2) {
        return hipErrorInvalidValue;
    }

    const int bin_count = edge_count - 1;
    std::fill(host_counts, host_counts + bin_count, 0ULL);
    std::fill(host_sums, host_sums + bin_count, 0.0F);
    if (sample_count > 0U && host_bin_numbers != nullptr) {
        std::fill(host_bin_numbers, host_bin_numbers + sample_count, 0U);
    }

    if (sample_count == 0) {
        return hipSuccess;
    }

    DeviceBuffer<float> d_samples;
    DeviceBuffer<float> d_values;
    DeviceBuffer<float> d_edges;
    DeviceBuffer<unsigned long long> d_counts;
    DeviceBuffer<float> d_sums;
    DeviceBuffer<unsigned int> d_bin_numbers;

    hipError_t status = d_samples.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_values.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_edges.allocate(edge_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_counts.allocate(bin_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_sums.allocate(bin_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_bin_numbers.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_samples.get(),
        host_samples,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_values.get(),
        host_values,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_edges.get(),
        host_edges,
        edge_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemset(
        d_counts.get(),
        0,
        static_cast<std::size_t>(bin_count) * sizeof(unsigned long long)
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemset(
        d_sums.get(),
        0,
        static_cast<std::size_t>(bin_count) * sizeof(float)
    );
    if (status != hipSuccess) {
        return status;
    }

    status = launch_binned_statistic_1d(
        d_samples.get(),
        d_values.get(),
        sample_count,
        d_edges.get(),
        edge_count,
        d_counts.get(),
        d_sums.get(),
        d_bin_numbers.get()
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_counts,
        d_counts.get(),
        static_cast<std::size_t>(bin_count) * sizeof(unsigned long long),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_sums,
        d_sums.get(),
        static_cast<std::size_t>(bin_count) * sizeof(float),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_bin_numbers,
        d_bin_numbers.get(),
        sample_count * sizeof(unsigned int),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

hipError_t binned_statistic_2d(
    const float* host_x,
    const float* host_y,
    const float* host_values,
    std::size_t sample_count,
    const float* host_x_edges,
    int x_edge_count,
    const float* host_y_edges,
    int y_edge_count,
    unsigned long long* host_counts,
    float* host_sums,
    unsigned int* host_bin_numbers_x,
    unsigned int* host_bin_numbers_y
) noexcept {
    if (x_edge_count < 2 || y_edge_count < 2) {
        return hipErrorInvalidValue;
    }

    const int x_bin_count = x_edge_count - 1;
    const int y_bin_count = y_edge_count - 1;
    const std::size_t total_bins =
        static_cast<std::size_t>(x_bin_count)
        * static_cast<std::size_t>(y_bin_count);

    std::fill(host_counts, host_counts + total_bins, 0ULL);
    std::fill(host_sums, host_sums + total_bins, 0.0F);
    if (sample_count > 0U) {
        if (host_bin_numbers_x != nullptr) {
            std::fill(host_bin_numbers_x, host_bin_numbers_x + sample_count, 0U);
        }
        if (host_bin_numbers_y != nullptr) {
            std::fill(host_bin_numbers_y, host_bin_numbers_y + sample_count, 0U);
        }
    }

    if (sample_count == 0) {
        return hipSuccess;
    }

    DeviceBuffer<float> d_x;
    DeviceBuffer<float> d_y;
    DeviceBuffer<float> d_values;
    DeviceBuffer<float> d_x_edges;
    DeviceBuffer<float> d_y_edges;
    DeviceBuffer<unsigned long long> d_counts;
    DeviceBuffer<float> d_sums;
    DeviceBuffer<unsigned int> d_bin_numbers_x;
    DeviceBuffer<unsigned int> d_bin_numbers_y;

    hipError_t status = d_x.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_y.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_values.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_x_edges.allocate(x_edge_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_y_edges.allocate(y_edge_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_counts.allocate(total_bins);
    if (status != hipSuccess) {
        return status;
    }

    status = d_sums.allocate(total_bins);
    if (status != hipSuccess) {
        return status;
    }

    status = d_bin_numbers_x.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = d_bin_numbers_y.allocate(sample_count);
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_x.get(),
        host_x,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_y.get(),
        host_y,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_values.get(),
        host_values,
        sample_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_x_edges.get(),
        host_x_edges,
        x_edge_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        d_y_edges.get(),
        host_y_edges,
        y_edge_count * sizeof(float),
        hipMemcpyHostToDevice
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemset(
        d_counts.get(),
        0,
        total_bins * sizeof(unsigned long long)
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemset(
        d_sums.get(),
        0,
        total_bins * sizeof(float)
    );
    if (status != hipSuccess) {
        return status;
    }

    status = launch_binned_statistic_2d(
        d_x.get(),
        d_y.get(),
        d_values.get(),
        sample_count,
        d_x_edges.get(),
        x_edge_count,
        d_y_edges.get(),
        y_edge_count,
        d_counts.get(),
        d_sums.get(),
        d_bin_numbers_x.get(),
        d_bin_numbers_y.get()
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_counts,
        d_counts.get(),
        total_bins * sizeof(unsigned long long),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_sums,
        d_sums.get(),
        total_bins * sizeof(float),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_bin_numbers_x,
        d_bin_numbers_x.get(),
        sample_count * sizeof(unsigned int),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    status = hipMemcpy(
        host_bin_numbers_y,
        d_bin_numbers_y.get(),
        sample_count * sizeof(unsigned int),
        hipMemcpyDeviceToHost
    );
    if (status != hipSuccess) {
        return status;
    }

    return hipSuccess;
}

}  // namespace binstatcuda
